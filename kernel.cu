
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void histogram_kernel(unsigned int *buffer, unsigned int size, unsigned int histogram_size, unsigned int *output) 
{

    extern __shared__ int histogram_prv[];
    unsigned int m = (histogram_size - 1 / blockDim.x) + 1; 
  
    if (threadIdx.x < histogram_size) 
    {
      for (unsigned int j = 0; j < m && (threadIdx.x + (j)*blockDim.x) < histogram_size; j++)
        histogram_prv[threadIdx.x + j * blockDim.x] = 0;
    }
    __syncthreads();
  
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int str = blockDim.x * gridDim.x;
    while (i < size) 
    {
      atomicAdd(&(histogram_prv[buffer[i]]), 1);
      i += str;
    }
    __syncthreads();
  
    if (threadIdx.x < histogram_size) 
    {
      for (unsigned int j = 0; j <= m && (threadIdx.x + (j)*blockDim.x) < histogram_size; j++)
        atomicAdd(&(output[threadIdx.x + j * blockDim.x]),
        histogram_prv[threadIdx.x + j * blockDim.x]);
    }
  }
  
  /******************************************************************************
  Setup and invoke your kernel(s) in this function. You may also allocate more
  GPU memory if you need to
  *******************************************************************************/
  void histogram(unsigned int *input, unsigned int *bins,
                 unsigned int num_elements, unsigned int num_bins) {
  
    // INSERT CODE HERE
  
    int BLOCK_SIZE = 512;
  
    dim3 dim_grid(((num_elements - 1) / BLOCK_SIZE) + 1, 1, 1);
    dim3 dim_block(BLOCK_SIZE, 1, 1);
  
    int histogram_prv_size = num_bins * (sizeof(int));
    histogram_kernel<<<dim_grid, dim_block, histogram_prv_size>>>(input, num_elements, num_bins, bins);

  }
  